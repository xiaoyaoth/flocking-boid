#include "hip/hip_runtime.h"
#include "../FlockingBoid/header.cuh"

class Dummy{
public:
	int *data;

	__device__ void foo(){
		const int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if(idx<AGENT_NO_D){
			data[idx] = idx;
			printf("%d ", data[idx]);
		}
	}

	Dummy(){
		hipMalloc((void**)&data, AGENT_NO*sizeof(int));
		getLastCudaError("Dummy hipMalloc");
	}
};

class DummyPack{
public:
	Dummy *dummy;

	DummyPack(){
		Dummy *dummy_h = new Dummy();
		hipMalloc((void**)&dummy, sizeof(Dummy));
		getLastCudaError("DummyPack hipMalloc");
		hipMemcpy(dummy, dummy_h, sizeof(Dummy), hipMemcpyHostToDevice);
		getLastCudaError("DummyPack hipMemcpy");
	}
};

__device__ Dummy *dummy;

__global__ void foo_kernel(Dummy *dummy){
	dummy->foo();
}

__global__ void test_dp_kernel(DummyPack *dp){
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < AGENT_NO_D)
		//printf("%d ", dp->dummy->data[idx]);
		dp->dummy->data[idx] = 1;
}

int test1(){
	Dummy *dummy_d;
	int *dummy_d_data;
	hipMalloc((void**)&dummy_d, sizeof(Dummy));
	getLastCudaError("hipMalloc dummy_d");
	hipMalloc((void**)&dummy_d_data, AGENT_NO*sizeof(int));
	getLastCudaError("hipMalloc dummy_d_data");
	//hipMemcpy(&dummy, &dummy_d, sizeof(dummy_d), hipMemcpyHostToDevice);
	//getLastCudaError("hipMemcpyToSymbol dummy");
	hipMemcpy(&dummy_d->data, &dummy_d_data, sizeof(dummy_d->data), hipMemcpyHostToDevice);
	getLastCudaError("hipMemcpyToSymbol dummy_data");

	foo_kernel<<<1, BLOCK_SIZE>>>(dummy_d);
	getLastCudaError("foo_kernel");

	
	Dummy *dummy_h = new Dummy();
	int *dummy_h_data = (int*)malloc(AGENT_NO*sizeof(int));
	hipMemcpy(dummy_h, dummy_d, sizeof(Dummy), hipMemcpyDeviceToHost);
	getLastCudaError("hipMemcpy dummy_h");
	hipMemcpy(dummy_h_data, dummy_h->data, AGENT_NO*sizeof(int), hipMemcpyDeviceToHost);
	getLastCudaError("hipMemcpy dummy_h_data");

	for (int i=0; i<AGENT_NO; i++)
		printf("%d ", dummy_h_data[i]);
	return 0;
}

int main2(){
	DummyPack *dp_h = new DummyPack();
	DummyPack *dp_d;
	hipMalloc((void**)&dp_d, sizeof(DummyPack));
	hipMemcpy(dp_d, dp_h, sizeof(DummyPack), hipMemcpyHostToDevice);

	test_dp_kernel<<<1, AGENT_NO>>>(dp_d);
	dp_h = new DummyPack();
	hipMemcpy(dp_h, dp_d, sizeof(DummyPack), hipMemcpyDeviceToHost);
	Dummy *d_h = new Dummy();
	hipMemcpy(d_h, dp_h->dummy, sizeof(Dummy), hipMemcpyDeviceToHost);
	int *data_h = (int*)malloc(AGENT_NO*sizeof(int));
	hipMemcpy(data_h, d_h->data, AGENT_NO*sizeof(int), hipMemcpyDeviceToHost);
	for(int i=0; i<AGENT_NO; i++)
		printf("%d ", data_h[i]);
	
	printf("test");
	return 0;
}