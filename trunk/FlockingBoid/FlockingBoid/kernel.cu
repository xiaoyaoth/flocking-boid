#include "hip/hip_runtime.h"
#include "gsimcore.cuh"
#include "gsimapp_boid.cuh"
#include <fstream>

__global__ void seeAllAgents(GModel *gm){
	GAgent *ag = gm->getScheduler()->obtainAgentPerThread();
	if (ag != NULL)
		ag->getAgId();
}

void initOnDevice(float *x_pos, float *y_pos){
	float *x_pos_h, *y_pos_h;
	x_pos_h = (float*)malloc(AGENT_NO*sizeof(float));
	y_pos_h = (float*)malloc(AGENT_NO*sizeof(float));

	std::ifstream fin("pos_data.txt.10240");
	std::string rec;

	char *cstr, *p;
	int i = 0;
	cstr = (char *)malloc(20 * sizeof(char));
	while (!fin.eof() && i<AGENT_NO) {
		std::getline(fin, rec);
		std::strcpy(cstr, rec.c_str());
		if(strcmp(cstr,"")==0)
			break;
		p=strtok(cstr, " ");
		x_pos_h[i] = atof(p);
		p=strtok(NULL, " ");
		y_pos_h[i] = atof(p);
		i++;
	}
	size_t floatDataSize = AGENT_NO*sizeof(float);
	hipMemcpy(x_pos, x_pos_h, floatDataSize, hipMemcpyHostToDevice);
	hipMemcpy(y_pos, y_pos_h, floatDataSize, hipMemcpyHostToDevice);
	cudaCheckErrors("initOnDevice");
}
__global__ void addAgentsOnDevice(GModel *gm, float *x_pos, float *y_pos){
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < AGENT_NO_D){
		PreyBoid *ag = new PreyBoid();
		//GAgent *ag = new GAgent();
		ag->loc.x = x_pos[idx];
		ag->loc.y = y_pos[idx];
		ag->time = 0;
		ag->rank = 0;
		gm->addToScheduler(ag, idx);
		gm->addToWorld(ag, idx);
	}
}

void test1(){
	int gSize = GRID_SIZE;
	printf("sizeof(GModel*): %d\n", sizeof(GModel*));
	printf("sizeof(void*):   %d\n", sizeof(void*));

	GModel *model_h = new GModel();
	model_h->allocOnDevice();
	GModel *model;
	hipMalloc((void**)&model, sizeof(GModel));
	hipMemcpy(model, model_h, sizeof(GModel), hipMemcpyHostToDevice);
	delete model_h;

	float *x_pos, *y_pos;
	size_t floatDataSize = AGENT_NO*sizeof(float);
	hipMalloc((void**)&x_pos, floatDataSize);
	hipMalloc((void**)&y_pos, floatDataSize);
	initOnDevice(x_pos, y_pos);
	addAgentsOnDevice<<<gSize, BLOCK_SIZE>>>(model, x_pos, y_pos);
	seeAllAgents<<<gSize, BLOCK_SIZE>>>(model);

	schUtil::scheduleRepeatingAllAgents<<<gSize, BLOCK_SIZE>>>(model);
	int steps;
	std::cout<<"steps: ";
	std::cin>>steps;
	for (int i = 0; i < steps; i++)
		schUtil::step<<<gSize, BLOCK_SIZE>>>(model);
}

void readConfig(){
	std::ifstream fin;
	fin.open("config.txt");
	std::string rec;
	char *cstr, *p;
	cstr = (char *)malloc(100 * sizeof(char));
	int temp;

	while (!fin.eof()) {
		std::getline(fin, rec);
		std::strcpy(cstr, rec.c_str());
		if(strcmp(cstr,"")==0)
			break;
		p=strtok(cstr, "=");
		if(strcmp(p, "AGENT_NO")==0){
			p=strtok(NULL, "=");
			AGENT_NO = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(AGENT_NO_D), &AGENT_NO, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "STEPS")==0){
			p=strtok(NULL, "=");
			STEPS = atoi(p);
		}
		if(strcmp(p, "CELL_NO")==0){
			p=strtok(NULL, "=");
			CELL_NO = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(CELL_NO_D), &CELL_NO, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "CELL_RESO")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(CELL_RESO), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "BOARDER_L")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_L), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "BOARDER_R")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_R), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "BOARDER_U")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_U), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "BOARDER_D")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(BOARDER_D), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
		if(strcmp(p, "XLENGTH")==0){
			p=strtok(NULL, "=");
			temp = atoi(p);
			hipMemcpyToSymbol(HIP_SYMBOL(XLENGTH), &temp, sizeof(int), 0, hipMemcpyHostToDevice);
		}
	}
	cudaCheckErrors("readConfig");
	free(cstr);
	fin.close();
}

int main(int argc, char *argv[]){
	readConfig();

	int gSize = GRID_SIZE;
	GModel *model_h = new GModel();
	model_h->allocOnDevice();
	GModel *model;
	hipMalloc((void**)&model, sizeof(GModel));
	hipMemcpy(model, model_h, sizeof(GModel), hipMemcpyHostToDevice);

	float *x_pos, *y_pos;
	size_t floatDataSize = AGENT_NO*sizeof(float);
	hipMalloc((void**)&x_pos, floatDataSize);
	hipMalloc((void**)&y_pos, floatDataSize);
	initOnDevice(x_pos, y_pos);

	printf("size taken by the one agent:%d and all agents: %d\n",
		sizeof(PreyBoid), AGENT_NO*sizeof(PreyBoid));
	addAgentsOnDevice<<<gSize, BLOCK_SIZE>>>(model, x_pos, y_pos);

	//schUtil::scheduleRepeatingAllAgents<<<1, BLOCK_SIZE>>>(model);
	cudaCheckErrors("before going into the big loop");
	printf("steps: %d\n", STEPS);

	for (int i=0; i<STEPS; i++){
		c2dUtil::genNeighbor(model);
		schUtil::step<<<gSize, BLOCK_SIZE>>>(model);
	}
	cudaCheckErrors("finished");
	system("PAUSE");
	return 0;
}

void backupcode1(){ //devRand
	int gSize = GRID_SIZE;
	float *devRandDebug;
	hipMalloc((void**)&devRandDebug, STRIP*gSize*BLOCK_SIZE*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(randDebug), &devRandDebug, sizeof(devRandDebug),
		0, hipMemcpyHostToDevice);

	std::fstream randDebugOut;
	std::fstream randDebugOut2;
	randDebugOut.open("randDebugOut.txt", std::ios::out);
	randDebugOut2.open("randDebugOut2.txt", std::ios::out);
	float *hostRandDebug = (float*)malloc(STRIP*gSize*BLOCK_SIZE*sizeof(float));

	hipMemcpy(hostRandDebug, devRandDebug, 
		STRIP*gSize*BLOCK_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0; i<gSize*BLOCK_SIZE; i++) {
		randDebugOut2<<hostRandDebug[STRIP*i]<<"\t";
		randDebugOut2.flush();
	}
	randDebugOut2<<std::endl;
	for(int i=0; i<gSize*BLOCK_SIZE; i++) {
		randDebugOut2<<hostRandDebug[STRIP*i+1]<<"\t";
		randDebugOut2.flush();
	}
	randDebugOut2<<std::endl;
	for(int i=0; i<gSize*BLOCK_SIZE; i++) {
		randDebugOut<<
			hostRandDebug[STRIP*i]<<" \t"<<
			hostRandDebug[STRIP*i+1]<<" \t"<<
			hostRandDebug[STRIP*i+2]<<" \t"<<
			hostRandDebug[STRIP*i+3]<<" \t"<<
			std::endl;
		randDebugOut.flush();
	}
	randDebugOut.close();
}