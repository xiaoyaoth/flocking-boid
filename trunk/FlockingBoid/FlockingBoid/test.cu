#include "hip/hip_runtime.h"
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <cstring>

#ifndef gpuAssert
#include <stdio.h>
#define gpuAssert( condition ) { \
	if( (condition) != 0 ) { \
		fprintf( stderr, "\n FAILURE %s in %s, line %d\n", \
			hipGetErrorString(condition), __FILE__, __LINE__ ) ; \
		exit( 1 ); } }
#endif

enum boid_type {PREY_BOID, PREDATOR_BOID, FOOD_BOID};

__device__ float* data;

__global__ void test_kernel()
{
	const int tid = threadIdx.x;
	data[tid]=tid;
}

int main1(void)
{
	boid_type type = PREY_BOID;
	printf("enum: %d\n", type);
	const int NUMBER = 1024;
	const size_t size = NUMBER*sizeof(float);
	
	float *data_h = (float*)malloc(size);

	float *data_d;
	float *data_symbol;
	hipMalloc(&data_d, size);

	//gpuAssert( hipGetSymbolAddress((void**)&data_symbol, "data"));
	gpuAssert( hipMemcpyToSymbol(HIP_SYMBOL( data), &data_d, sizeof(data_d), 0, hipMemcpyHostToDevice ) );

	test_kernel<<<1, 1024>>>();

	gpuAssert( hipMemcpy( data_h, data_d, size, hipMemcpyDeviceToHost ) );
	for (int i = 0; i < NUMBER; i++)
		printf("%f ", data_h[i]);
	
	return (int)hipDeviceReset();
	
}