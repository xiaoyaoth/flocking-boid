#include "hip/hip_runtime.h"
#include "testheader.cuh"

class GModel;
class Continuous2D{
private:
	int *neighborIdx, *cellIdx;
public:
	void allocOnDevice(){
		size_t sizeAgArray = AGENT_NO*sizeof(int);
		size_t sizeCellArray = CELL_NO*sizeof(int);

		hipMalloc((void**)&neighborIdx, sizeAgArray);
		getLastCudaError("Continuous2D():hipMalloc:neighborIdx");
		hipMalloc((void**)&cellIdx, sizeCellArray);
		getLastCudaError("Continuous2D():hipMalloc:cellIdx");


		int *neighborIdx_h = (int*)malloc(sizeAgArray);
		int *cellIdx_h = (int*)malloc(sizeCellArray);
		for(int i=0; i<AGENT_NO; i++)
			neighborIdx_h[i] = 2;
		for(int i = 0; i<CELL_NO; i++)
			cellIdx_h[i] = 3;
		hipMemcpy(this->neighborIdx, neighborIdx_h, sizeAgArray, hipMemcpyHostToDevice);
		getLastCudaError("Continuous2D():hipMemcpy:neighborIdx");
		hipMemcpy(this->cellIdx, cellIdx_h, sizeCellArray, hipMemcpyHostToDevice);
		getLastCudaError("Continuous2D():hipMemcpy:cellIdx");
	}
	void allocOnHost(){
		size_t sizeAgArray = AGENT_NO*sizeof(int);
		size_t sizeCellArray = CELL_NO*sizeof(int);
		neighborIdx = (int*)malloc(sizeAgArray);
		cellIdx = (int*)malloc(sizeCellArray);
	}
	friend void seeThrough(GModel *model);
	friend void test1();
};

class GModel{
private:
	Continuous2D *world;
public:
	void allocOnDevice(){
		Continuous2D *world_d = new Continuous2D();
		world_d->allocOnDevice();
		hipMalloc((void**)&world, sizeof(Continuous2D));
		getLastCudaError("GModel():hipMalloc:world");
		hipMemcpy(world, world_d, sizeof(Continuous2D), hipMemcpyHostToDevice);
		getLastCudaError("GModel():hipMemcpy:world");
	}
	void allocOnHost(){
		world = new Continuous2D();
		world->allocOnHost();
	}
	friend void seeThrough(GModel *model);
	friend void test1();
};

__global__ void seeThrough(GModel *model){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < AGENT_NO){
		int a = model->world->neighborIdx[idx];
		int b = model->world->cellIdx[idx];
	}
}

void test1(){
	size_t sizeAgArray = AGENT_NO*sizeof(int);
	size_t sizeCellArray = CELL_NO*sizeof(int);

	GModel *model_h = new GModel();
	model_h->allocOnDevice();
	GModel *model_d;
	hipMalloc((void**)&model_d, sizeof(GModel));
	hipMemcpy(model_d, model_h, sizeof(GModel), hipMemcpyHostToDevice);
	delete model_h;
	seeThrough<<<1, BLOCK_SIZE>>>(model_d);
	model_h = new GModel();
	hipMemcpy(model_h, model_d, sizeof(GModel), hipMemcpyDeviceToHost);
	Continuous2D *world_h = new Continuous2D();
	hipMemcpy(world_h, model_h->world, sizeof(Continuous2D), hipMemcpyDeviceToHost);
	int *data_h = (int*)malloc(sizeAgArray);
	int *data2_h = (int*)malloc(sizeCellArray);
	hipMemcpy(data_h, world_h->neighborIdx, sizeAgArray, hipMemcpyDeviceToHost);
	hipMemcpy(data2_h, world_h->cellIdx, sizeCellArray, hipMemcpyDeviceToHost);
	for (int i=0; i<AGENT_NO; i++)
		printf("%d ", data_h[i]);
	for (int i=0; i<CELL_NO; i++)
		printf("%d ", data2_h[i]);

	printf("\nDONE");

}

int main3(){
	test1();
	return 0;
}

