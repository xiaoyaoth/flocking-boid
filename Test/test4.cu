
#include<stdlib.h>
#include<time.h>
#include<stdio.h>
#include"hip/hip_runtime.h"

#include"hiprand/hiprand.h"
#include"hiprand/hiprand_kernel.h"

#define HOSTGEN	1
#define DEVGEN	0
#define BLOCK_SIZE	1024
#define	CUDA_CALL(x)do{if((x)!=hipSuccess){\
	printf("Errorat%s:%d\n",__FILE__,__LINE__);\
	return EXIT_FAILURE;}}while(0)

class RNG{
	hiprandState* states;
	__device__ float nextFloat(){
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		return hiprand_uniform(&states[idx]);
	}
	friend __global__ void initStates(RNG *rng);
};

__global__ void initStates(RNG *rng){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
}

__global__ void setup_kernel(hiprandState*state){
	int id=threadIdx.x+blockIdx.x*64;
	/*Eachthreadgetssameseed,adifferentsequence
	number,nooffset*/
	hiprand_init(1234,id,0,&state[id]);
}

__global__ void generate_kernel(hiprandState*state,int*result){
	int id=threadIdx.x+blockIdx.x*64;
	int count=0;
	unsigned int x;
	/*Copystatetolocalmemoryforefficiency*/
	hiprandState localState=state[id];
	/*Generatepseudo-randomunsignedints*/
	//for(int n=0; n<100000; n++){
		x=hiprand(&localState);
		/*Checkiflowbitset*/
		if(x&1){
			count++;
		}
	//}
	/*Copystatebacktoglobalmemory*/
	state[id]=localState;
	/*Storeresults*/
	result[id]+=count;
}

__global__ void setZero(int* res){
	int idx = threadIdx.x + blockIdx.x * 64;
	res[idx] = 0;
}

__global__ void consume(int *res, unsigned int *pool){
	int idx = threadIdx.x + blockIdx.x * 64;
	int x = pool[idx];
	if (x&1)
		res[idx]+=1;
}

int test4_1()
{
	int i,total;
	hiprandState*devStates;
	int*devResults,*hostResults;
	/*Allocatespaceforresultsonhost*/
	hostResults=(int*)calloc(64*64,sizeof(int));
	/*Allocatespaceforresultsondevice*/
	CUDA_CALL(hipMalloc((void**)&devResults,64*64*sizeof(int)));
	/*Setresultsto0*/
	CUDA_CALL(hipMemset(devResults,0,64*64*sizeof(int)));
	/*Allocatespaceforprngstatesondevice*/
	CUDA_CALL(hipMalloc((void**)&devStates,64*64*sizeof(hiprandState)));
	/*Setupprngstates*/
	setup_kernel<<<64,64>>>(devStates);
	/*Generateandusepseudo-random*/
	for(i=0;i<10000;i++){
		generate_kernel<<<64,64>>>(devStates,devResults);
	}
	/*Copydevicememorytohost*/
	CUDA_CALL(hipMemcpy(hostResults,devResults,64*64*
		sizeof(int),hipMemcpyDeviceToHost));
	/*Showresult*/
	total=0;
	for(i=0;i<64*64;i++){
		total+=hostResults[i];
	}
	printf("Fraction with low bit set was %10.13f\n",
		(float)total/(64.0f*64.0f*10000.0f));
	/*Cleanup*/
	CUDA_CALL(hipFree(devStates));
	CUDA_CALL(hipFree(devResults));
	free(hostResults);
	return EXIT_SUCCESS;
}

int test4_2(){
	unsigned int *pool;
	int *res;
	int *res_h;

	hipMalloc(&pool,64*64*sizeof(int));
	hipMalloc(&res, 64*64*sizeof(int));
	hipMemset(res, 0, 64*64*sizeof(int));
	res_h = (int*)calloc(64*64,sizeof(int));

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,time(0));

	for(int i=0;i<10000;i++){
		hiprandGenerate(gen, pool, 64*64);
		consume<<<64,64>>>(res, pool);
	}

	CUDA_CALL(hipMemcpy(res_h,res,64*64*sizeof(int),hipMemcpyDeviceToHost));

	int total=0;
	for(int i=0;i<64*64;i++){
		total+=res_h[i];
	}
	printf("Fraction with low bit set was %10.13f\n",
		(float)total/(64.0f*64.0f*10000.0f));
	/*Cleanup*/
	CUDA_CALL(hipFree(pool));
	CUDA_CALL(hipFree(res));
	free(res_h);
	return EXIT_SUCCESS;
}

int main(int argc, char** argv){
	int sel = atoi(argv[1]);
	if(sel == 1)
		test4_1();
	else
		test4_2();
	system("PAUSE");
	return EXIT_SUCCESS;
}